#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_kernels.h"

__global__ void vectorAddKernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void vectorAdd(const float* a, const float* b, float* c, int n) {
    // For CPU-only systems, do simple CPU addition
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        // CPU fallback
        for (int i = 0; i < n; i++) {
            c[i] = a[i] + b[i];
        }
        return;
    }
    
    // GPU implementation
    float *d_a, *d_b, *d_c;
    size_t size = n * sizeof(float);
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloKernel() {
    printf("Hello from CUDA kernel! Thread %d of Block %d\n", 
           threadIdx.x, blockIdx.x);
}

int main() {
    std::cout << "CUDA Hello World Test" << std::endl;
    std::cout << "====================" << std::endl;
    
    // Get device properties
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found. Running CPU-only test." << std::endl;
        std::cout << "CUDA compilation test successful!" << std::endl;
        return 0;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    // Launch kernel
    helloKernel<<<2, 4>>>();
    hipDeviceSynchronize();
    
    std::cout << "CUDA test successful!" << std::endl;
    
    return 0;
}